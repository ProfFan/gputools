
#include <hip/hip_runtime.h>
#define max(a, b) ((a > b)?a:b)

#define THREADSPERDIM   16

#define FALSE 0
#define TRUE !FALSE

// mX has order rows x cols
// vectY has length rows
__global__ void getRestricted(int countx, int county, int rows, int cols, 
                              float * mX, int mXdim, float * vY, int vYdim, float * mQ, int mQdim,
                              float * mR, int mRdim, float * vectB, int vectBdim) {

  int 
    m = blockIdx.x * THREADSPERDIM + threadIdx.x, n,
    i, j, k;
  float 
    sum, invnorm,
    * X, * Y, * Q, * R, * B,
    * coli, * colj, 
    * colQ, * colX;

  if(m >= county) return;
  if(m == 1) n = 0;
  else n = 1;

  X = mX + (m * mXdim);
  // initialize the intercepts
  for(i = 0; i < rows; i++)
    X[i] = 1.f;

  Y = vY + (m * countx + n) * vYdim;
  B = vectB + m * vectBdim;
  Q = mQ + m * mQdim;
  R = mR + m * mRdim;

  // initialize Q with X ...
  for(i = 0; i < rows; i++) {
    for(j = 0; j < cols; j++)
      Q[i+j*rows] = X[i+j*rows];
  }

  // gramm-schmidt process to find Q
  for(j = 0; j < cols; j++) {
    colj = Q+rows*j;
    for(i = 0; i < j; i++) {
      coli = Q+rows*i;
      sum = 0.f;
      for(k = 0; k < rows; k++)
        sum += coli[k] * colj[k];
      for(k = 0; k < rows; k++)
        colj[k] -= sum * coli[k];
    }
    sum = 0.f;
    for(i = 0; i < rows; i++)
      sum += colj[i] * colj[i];
    invnorm = 1.f / sqrtf(sum);
    for(i = 0; i < rows; i++)
      colj[i] *= invnorm;
  }
  for(i = cols-1; i > -1; i--) {
    colQ = Q+i*rows;
    // matmult Q * X -> R
    for(j = 0; j < cols; j++) {
      colX = X+j*rows;
      sum = 0.f;
      for(k = 0; k < rows; k++)
        sum += colQ[k] * colX[k];
      R[i+j*cols] = sum;
    }
    sum = 0.f;
    // compute the vector Q^t * Y -> B
    for(j = 0; j < rows; j++) 
      sum += colQ[j] * Y[j];
    // back substitution to find the x for Rx = B
    for(j = cols-1; j > i; j--)
      sum -= R[i+j*cols] * B[j];

    B[i] = sum / R[i+i*cols];
  }
}

// mX has order rows x cols
// vectY has length rows
__global__ void getUnrestricted(int countx, int county, int rows, int cols, 
                                float * mX, int mXdim, float * vY, int vYdim, float * mQ, int mQdim,
                                float * mR, int mRdim, float * vectB, int vectBdim) {

  int 
    n = blockIdx.x * THREADSPERDIM + threadIdx.x, 
    m = blockIdx.y * THREADSPERDIM + threadIdx.y, 
    i, j, k;
  float 
    sum, invnorm,
    * X, * Y, * Q, * R, * B,
    * coli, * colj, 
    * colQ, * colX;
  if((m >= county) || (n >= countx)) return;

  X = mX + (m * countx + n) * mXdim;
  // initialize the intercepts
  for(i = 0; i < rows; i++) 
    X[i] = 1.f;

  Y = vY + (m*countx+n) * vYdim;
  B = vectB + (m*countx+n) * vectBdim;
  Q = mQ + (m*countx+n) * mQdim;
  R = mR + (m*countx+n) * mRdim;

  // initialize Q with X ...
  for(i = 0; i < rows; i++) {
    for(j = 0; j < cols; j++)
      Q[i+j*rows] = X[i+j*rows];
  }

  // gramm-schmidt process to find Q
  for(j = 0; j < cols; j++) {
    colj = Q+rows*j;
    for(i = 0; i < j; i++) {
      coli = Q+rows*i;
      sum = 0.f;
      for(k = 0; k < rows; k++)
        sum += coli[k] * colj[k];
      for(k = 0; k < rows; k++)
        colj[k] -= sum * coli[k];
    }
    sum = 0.f;
    for(i = 0; i < rows; i++)
      sum += colj[i] * colj[i];
    invnorm = 1.f / sqrtf(sum);
    for(i = 0; i < rows; i++)
      colj[i] *= invnorm;
  }
  for(i = cols-1; i > -1; i--) {
    colQ = Q+i*rows;
    // matmult Q * X -> R
    for(j = 0; j < cols; j++) {
      colX = X+j*rows;
      sum = 0.f;
      for(k = 0; k < rows; k++)
        sum += colQ[k] * colX[k];
      R[i+j*cols] = sum;
    }
    sum = 0.f;
    // compute the vector Q^t * Y -> B
    for(j = 0; j < rows; j++) 
      sum += colQ[j] * Y[j];
    // back substitution to find the x for Rx = B
    for(j = cols-1; j > i; j--)
      sum -= R[i+j*cols] * B[j];

    B[i] = sum / R[i+i*cols];
  }
}

__global__ void ftest(int diagFlag, int p, int rows, int colsx, int colsy, 
                      int rCols, int unrCols, float * obs, int obsDim, 
                      float * rCoeffs, int rCoeffsDim, float * unrCoeffs, int unrCoeffsDim, 
                      float * rdata, int rdataDim, float * unrdata, int unrdataDim, 
                      float * dfStats) // float * dpValues)
{
  int 
    j = blockIdx.x * THREADSPERDIM + threadIdx.x, 
    i = blockIdx.y * THREADSPERDIM + threadIdx.y, 
    idx = i*colsx + j, k, m;
  float 
    kobs, fp = (float) p, frows = (float) rows,
    rSsq, unrSsq,
    rEst, unrEst,
    score = 0.f, 
    * tObs, * tRCoeffs, * tUnrCoeffs, 
    * tRdata, * tUnrdata; 

  if((i >= colsy) || (j >= colsx)) return;
  if((!diagFlag) && (i == j)) {
    dfStats[idx] = 0.f;
    // dpValues[idx] = 0.f;
    return;
  }

  tObs = obs + (i*colsx+j)*obsDim;

  tRCoeffs = rCoeffs + i*rCoeffsDim;
  tRdata = rdata + i*rdataDim;
        
  tUnrCoeffs = unrCoeffs + (i*colsx+j)*unrCoeffsDim;
  tUnrdata = unrdata + (i*colsx+j)*unrdataDim;

  rSsq = unrSsq = 0.f;
  for(k = 0; k < rows; k++) {
    unrEst = rEst = 0.f;
    kobs = tObs[k];
    for(m = 0; m < rCols; m++)
      rEst += tRCoeffs[m] * tRdata[k+m*rows];
    for(m = 0; m < unrCols; m++) 
      unrEst += tUnrCoeffs[m] * tUnrdata[k+m*rows];
    rSsq   += (kobs - rEst) * (kobs - rEst);
    unrSsq += (kobs - unrEst) * (kobs - unrEst);

  }
  score = ((rSsq - unrSsq)*(frows-2.f*fp-1.f)) / (fp*unrSsq);

  if(!isfinite(score))
    score = 0.f;

  dfStats[idx] = score;
}
